#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "env_param.hh"
//#include "scat_param.hh"
#include "mol_param.hh"
#include "WaasKirf.hh"
#define PI 3.14159265359
//#include <hip/hip_fp16.h>


__global__ void dist_calc (
    float *coord, 
    int *close_num,
    int *close_flag, 
    int *close_idx, 
    int num_atom, 
    int num_atom2) {

    // close_flag is a 1024 x num_atom2 int matrix initialized to 0.
    // close_idx: A num_atom x 200 int matrix, row i of which only the first close_num[i] elements are defined. (Otherwise it's -1). 
    __shared__ float x_ref, y_ref, z_ref;
    __shared__ int idz;
    __shared__ int temp[2048];
    // Calc distance
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        if (threadIdx.x == 0) {
            x_ref = coord[3*ii  ];
            y_ref = coord[3*ii+1];
            z_ref = coord[3*ii+2];
        }
        int idy = ii % gridDim.x; // This will be what row of close_flag this block is putting its value in.
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            float r2t = (coord[3*jj  ] - x_ref) * (coord[3*jj  ] - x_ref) + 
                        (coord[3*jj+1] - y_ref) * (coord[3*jj+1] - y_ref) + 
                        (coord[3*jj+2] - z_ref) * (coord[3*jj+2] - z_ref); 
 
            if (r2t < 34.0) {
                close_flag[idy*num_atom2+jj] = 1; // roughly 2 A + 2 A vdW + 2 * 1.8 A probe
            } else { 
                close_flag[idy*num_atom2+jj] = 0;
            }
            if (ii == jj) close_flag[idy*num_atom2+jj] = 0;
        }
        __syncthreads();
        // Do pre scan
        idz = 0;
        int temp_sum = 0;
        for (int jj = threadIdx.x; jj < num_atom2; jj += 2 * blockDim.x) {
            int idx = jj % blockDim.x; 
            int offset = 1;
            temp[2 * idx]     = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx];
            temp[2 * idx + 1] = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1];
            for (int d = 2 * blockDim.x>>1; d > 0; d >>= 1) { // up-sweep
                __syncthreads();
                if (idx < d) {
                    int ai = offset * (2 * idx + 1) - 1;
                    int bi = offset * (2 * idx + 2) - 1;
                    temp[bi] += temp[ai];
                }
                offset *= 2;
            }
            __syncthreads();
            temp_sum = close_num[ii];
            __syncthreads();
            if (idx == 0) {
                close_num[ii] += temp[2 * blockDim.x - 1]; // log the total number of 1's in this blockDim
                temp[2 * blockDim.x - 1] = 0;
            }
            __syncthreads();
            for (int d = 1; d < blockDim.x * 2; d *= 2) { //down-sweep
                offset >>= 1;
                __syncthreads();
                if (idx < d) {
                    int ai = offset * (2 * idx + 1) - 1;
                    int bi = offset * (2 * idx + 2) - 1;
                    int t    = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
        
            __syncthreads();
        
            // Finally assign the indices
            if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx] == 1) {
                close_idx[ii * 1024 + temp[2*idx] + temp_sum] = 2 * idx + 2 * blockDim.x * idz;
            }
            if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1] == 1) {
                close_idx[ii * 1024 + temp[2*idx+1] + temp_sum] = 2*idx+1 + 2 * blockDim.x * idz;
            }
            idz++;
            __syncthreads();
        }
    }
}

__global__ void __launch_bounds__(512,4) surf_calc_surf_grad (
    float *coord, 
    int *Ele, 
    int *close_num, 
    int *close_idx, 
    float *vdW, 
    int num_atom, 
    int num_atom2, 
    int num_raster, 
    float sol_s, 
    float *V,
    float *surf_grad,
    float offset) {

    // num_raster should be a number of 2^n. 
    // sol_s is solvent radius (default = 1.8 A)
    __shared__ float vdW_s; // vdW radius of the center atom
    __shared__ int pts[512]; // All spherical raster points
    __shared__ int ptspx[512], ptsmx[512], ptspy[512], ptsmy[512], ptspz[512], ptsmz[512];
    __shared__ float L, r;
    
    if (blockIdx.x >= num_atom) return;
    L = sqrt(num_raster * PI);
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        int atom1t = Ele[ii];
        if (atom1t > 5) atom1t = 0;
        vdW_s = vdW[atom1t];
        r = vdW_s + sol_s;
        for (int jj = threadIdx.x; jj < num_raster; jj += blockDim.x) {
            int pt = 1;
            int ptpx = 1;
            int ptmx = 1;
            int ptpy = 1;
            int ptmy = 1;
            int ptpz = 1;
            int ptmz = 1;
            
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // vdW points
            float x = vdW_s * xu + coord[3*ii];
            float y = vdW_s * yu + coord[3*ii+1];
            float z = vdW_s * zu + coord[3*ii+2];
            // Solvent center
            float x2 = r * xu + coord[3*ii];
            float y2 = r * yu + coord[3*ii+1];
            float z2 = r * zu + coord[3*ii+2];
            for (int kk = 0; kk < close_num[ii]; kk++) {
                int atom2i = close_idx[ii * 1024 + kk];
                int atom2t = Ele[atom2i];
                if (atom2t > 5) atom2t = 0;
                float dx = (x - coord[3*atom2i]);
                float dy = (y - coord[3*atom2i+1]);
                float dz = (z - coord[3*atom2i+2]);
                float dx2 = (x2 - coord[3*atom2i]);
                float dy2 = (y2 - coord[3*atom2i+1]);
                float dz2 = (z2 - coord[3*atom2i+2]);
                float dr2 = dx * dx + dy * dy + dz * dz; 
                float dr22 = dx2 * dx2 + dy2 * dy2 + dz2 * dz2;
                // vdW points must not cross into other atom
                if (dr2 < vdW[atom2t] * vdW[atom2t]) pt = 0; //pts[jj] = 0;
                // solvent center has to be far enough
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) pt = 0; //pts[jj] = 0;

                // Plus x
                dr2 =  (dx + offset)  * (dx + offset) + dy * dy + dz * dz;
                dr22 = (dx2 + offset) * (dx2 + offset) + dy2 * dy2 + dz2 * dz2;
                if (dr2 < vdW[atom2t] * vdW[atom2t]) ptpx = 0; //ptspx[jj] = 0;
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) ptpx = 0; //ptspx[jj] = 0;
                // Minus x
                dr2 =  (dx - offset)  * (dx - offset) + dy * dy + dz * dz;
                dr22 = (dx2 - offset) * (dx2 - offset) + dy2 * dy2 + dz2 * dz2;
                if (dr2 < vdW[atom2t] * vdW[atom2t]) ptmx = 0; //ptsmx[jj] = 0;
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) ptmx = 0; //ptsmx[jj] = 0;
                // Plus y
                dr2 =  dx * dx   + (dy + offset)  * (dy + offset) + dz * dz; 
                dr22 = dx2 * dx2 + (dy2 + offset) * (dy2 + offset) + dz2 * dz2;
                if (dr2 < vdW[atom2t] * vdW[atom2t]) ptpy = 0; //ptspy[jj] = 0;
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) ptpy = 0; //ptspy[jj] = 0;
                // Minus y
                dr2 =  dx * dx   + (dy - offset)  * (dy - offset) + dz * dz; 
                dr22 = dx2 * dx2 + (dy2 - offset) * (dy2 - offset) + dz2 * dz2;
                if (dr2 < vdW[atom2t] * vdW[atom2t]) ptmy = 0; //ptsmy[jj] = 0;
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) ptmy = 0; //ptsmy[jj] = 0;
                // Plus z
                dr2 =  dx * dx + dy * dy + (dz + offset) * (dz + offset); 
                dr22 = dx2 * dx2 + dy2 * dy2 + (dz2 + offset) * (dz2 + offset);
                if (dr2 < vdW[atom2t] * vdW[atom2t]) ptpz = 0; //ptspz[jj] = 0;
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) ptpz = 0; //ptspz[jj] = 0;
                // Minus z
                dr2 =  dx * dx + dy * dy + (dz - offset) * (dz - offset); 
                dr22 = dx2 * dx2 + dy2 * dy2 + (dz2 - offset) * (dz2 - offset);
                if (dr2 < vdW[atom2t] * vdW[atom2t]) ptmz = 0; //ptsmz[jj] = 0;
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) ptmz = 0; //ptsmz[jj] = 0;
                
            }
            pts[jj] = pt;
            ptspx[jj] = ptpx;
            ptsmx[jj] = ptmx;
            ptspy[jj] = ptpy;
            ptsmy[jj] = ptmy;
            ptspz[jj] = ptpz;
            ptsmz[jj] = ptmz;
        }
        // Sum pts == 1, calc surf area and assign to V[ii]
        for (int stride = num_raster / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                pts[iAccum] += pts[stride + iAccum];

                ptspx[iAccum] += ptspx[stride + iAccum];
                ptsmx[iAccum] += ptsmx[stride + iAccum];
                ptspy[iAccum] += ptspy[stride + iAccum];
                ptsmy[iAccum] += ptsmy[stride + iAccum];
                ptspz[iAccum] += ptspz[stride + iAccum];
                ptsmz[iAccum] += ptsmz[stride + iAccum];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            V[ii] = (float)pts[0]/(float)num_raster;// * 4.0 * r * r * PI ;

            surf_grad[3*ii  ] = (float)(ptspx[0] - ptsmx[0]) / 2.0 / offset / (float)num_raster;
            surf_grad[3*ii+1] = (float)(ptspy[0] - ptsmy[0]) / 2.0 / offset / (float)num_raster;
            surf_grad[3*ii+2] = (float)(ptspz[0] - ptsmz[0]) / 2.0 / offset / (float)num_raster;
        }
    }
}

__global__ void __launch_bounds__(512,4) surf_calc (
    float *coord, 
    int *Ele, 
    int *close_num, 
    int *close_idx, 
    float *vdW, 
    int num_atom, 
    int num_atom2, 
    int num_raster, 
    float sol_s, 
    float *V) {

    // num_raster should be a number of 2^n. 
    // sol_s is solvent radius (default = 1.8 A)
    __shared__ float vdW_s; // vdW radius of the center atom
    __shared__ int pts[512]; // All spherical raster points
    __shared__ float L, r;
    
    if (blockIdx.x >= num_atom) return;
    L = sqrt(num_raster * PI);
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        int atom1t = Ele[ii];
        if (atom1t > 5) atom1t = 0;
        vdW_s = vdW[atom1t];
        r = vdW_s + sol_s;
        for (int jj = threadIdx.x; jj < num_raster; jj += blockDim.x) {
            int pt = 1;
            
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // vdW points
            float x = vdW_s * xu + coord[3*ii];
            float y = vdW_s * yu + coord[3*ii+1];
            float z = vdW_s * zu + coord[3*ii+2];
            // Solvent center
            float x2 = r * xu + coord[3*ii];
            float y2 = r * yu + coord[3*ii+1];
            float z2 = r * zu + coord[3*ii+2];
            for (int kk = 0; kk < close_num[ii]; kk++) {
                int atom2i = close_idx[ii * 1024 + kk];
                int atom2t = Ele[atom2i];
                if (atom2t > 5) atom2t = 0;
                float dx = (x - coord[3*atom2i]);
                float dy = (y - coord[3*atom2i+1]);
                float dz = (z - coord[3*atom2i+2]);
                float dx2 = (x2 - coord[3*atom2i]);
                float dy2 = (y2 - coord[3*atom2i+1]);
                float dz2 = (z2 - coord[3*atom2i+2]);
                float dr2 = dx * dx + dy * dy + dz * dz; 
                float dr22 = dx2 * dx2 + dy2 * dy2 + dz2 * dz2;
                // vdW points must not cross into other atom
                if (dr2 < vdW[atom2t] * vdW[atom2t]) pt = 0; //pts[jj] = 0;
                // solvent center has to be far enough
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) pt = 0; //pts[jj] = 0;
                
            }
            pts[jj] = pt;
        }
        // Sum pts == 1, calc surf area and assign to V[ii]
        for (int stride = num_raster / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                pts[iAccum] += pts[stride + iAccum];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            V[ii] = (float)pts[0]/(float)num_raster;// * 4.0 * r * r * PI ;
        }
    }
}


__global__ void sum_V (
    float *V, 
    float *V_s, 
    int num_atom, 
    int num_atom2, 
    int *Ele, 
    float *vdW) {

    for (int ii = threadIdx.x; ii < num_atom2; ii += blockDim.x) {
        if (ii < num_atom) {
            int atomi = Ele[ii];
            if (atomi > 5) atomi = 0;
            V_s[ii] = V[ii] * 4.0 * PI * vdW[atomi] * vdW[atomi];
        } else {
            V_s[ii] = 0.0;
        }
    }
    for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
            V_s[iAccum] += V_s[stride + iAccum];
        }
    }
    __syncthreads();
    if (threadIdx.x == 0) printf("Convex contact area = %.3f A^2.\n", V_s[0]);
}

__global__ void FF_calc (
    float *q_S_ref_dS, 
    float *WK, 
    float *vdW, 
    int num_q, 
    int num_ele, 
    float c1, 
    float r_m, 
    float *FF_table,
    float rho) {

    __shared__ float q_pt, q_WK, C1, expC1;
    __shared__ float FF_pt[7]; // num_ele + 1, the last one for water.
    __shared__ float vdW_s[7];
    __shared__ float WK_s[66]; 
    __shared__ float C1_PI_43_rho;
    if (blockIdx.x >= num_q) return; // out of q range
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        q_pt = q_S_ref_dS[ii];
        q_WK = q_pt / 4.0 / PI;
        // FoXS C1 term
        expC1 = -powf(4.0 * PI / 3.0, 1.5) * q_WK * q_WK * r_m * r_m * (c1 * c1 - 1.0) / 4.0 / PI;
        C1 = powf(c1,3) * exp(expC1);
        C1_PI_43_rho = C1 * PI * 4.0 / 3.0 * rho;
        for (int jj = threadIdx.x; jj < 11 * num_ele; jj += blockDim.x) {
            WK_s[jj] = WK[jj];
        }
        __syncthreads();

        // Calculate Form factor for this block (or q vector)
        for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
            vdW_s[jj] = vdW[jj];
            if (jj == num_ele) {
                // water
                FF_pt[jj] = WK_s[3*11+5];
                FF_pt[jj] += 2.0 * WK_s[5];
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);
                for (int kk = 0; kk < 5; kk ++) {
                    FF_pt[jj] += WK_s[3*11+kk] * exp(-WK_s[3*11+kk+6] * q_WK * q_WK);
                    FF_pt[jj] += WK_s[kk] * exp(-WK_s[kk+6] * q_WK * q_WK);
                    FF_pt[jj] += WK_s[kk] * exp(-WK_s[kk+6] * q_WK * q_WK);
                }
            } else { 
                FF_pt[jj] = WK_s[jj*11+5];
                // The part is for excluded volume
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);
                for (int kk = 0; kk < 5; kk++) {
                    FF_pt[jj] += WK_s[jj*11+kk] * exp(-WK_s[jj*11+kk+6] * q_WK * q_WK); 
                }
            }
            FF_table[ii*(num_ele+1)+jj] = FF_pt[jj];
        }
    }
}

__global__ void create_FF_full_HyPred (
    float *FF_table, 
    float *V, 
    float c2_F,
    float *c2_H,
    int *Ele, 
    float *FF_full, 
    int num_q, 
    int num_ele, 
    int num_atom, 
    int num_atom2) {
    
    __shared__ float FF_pt[7];
    __shared__ float hydration[10];
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        // Get form factor for this block (or q vector)
        if (ii < num_q) {
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
            }
        }
        __syncthreads();

        for (int jj = threadIdx.x; jj < 10; jj += blockDim.x) {
            hydration[jj] = c2_F * c2_H[jj] * FF_pt[num_ele];
        }
        __syncthreads();
        
        // Calculate atomic form factor for this q
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            int atomt = Ele[jj];
            if (atomt > 5) {  // Which means this is a hydrogen
                FF_full[ii*num_atom2 + jj] = FF_pt[0];
                FF_full[ii*num_atom2 + jj] += hydration[atomt] * V[jj];
            } else { // Heavy atoms - do the same as before
                FF_full[ii*num_atom2 + jj] = FF_pt[atomt];
                FF_full[ii*num_atom2 + jj] += hydration[atomt] * V[jj];
            }
        }
    }
}

__global__ void create_FF_full_FoXS (
    float *FF_table, 
    float *V, 
    float c2, 
    int *Ele, 
    float *FF_full, 
    int num_q, 
    int num_ele, 
    int num_atom, 
    int num_atom2) {

    __shared__ float FF_pt[7];
    float hydration;
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        // Get form factor for this block (or q vector)
        if (ii < num_q) {
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
            }
        }
        __syncthreads();
        
        // In FoXS since c2 remains the same for all elements it is reduced to one value.
        hydration = c2 * FF_pt[num_ele];
        
        // Calculate atomic form factor for this q
        // However to keep compatible to HyPred method we leave atom type def unchanged.
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            int atomt = Ele[jj];
            if (atomt > 5) {  // Which means this is a hydrogen
                FF_full[ii*num_atom2 + jj] = FF_pt[0];
                FF_full[ii*num_atom2 + jj] += hydration * V[jj];
            } else {          // Heavy atoms - do the same as before
                FF_full[ii*num_atom2 + jj] = FF_pt[atomt];
                FF_full[ii*num_atom2 + jj] += hydration * V[jj];
            }
        }
    }
}

__global__ void create_FF_full_FoXS_surf_grad (
    float *FF_table, 
    float *V, 
    float c2, 
    int *Ele, 
    float *FF_full,
    float *surf_grad, 
    int num_q, 
    int num_ele, 
    int num_atom, 
    int num_atom2) {

    __shared__ float FF_pt[7];
    float hydration;
    for (int ii = blockIdx.x; ii < num_q+1; ii += gridDim.x) {

        // Get form factor for this block (or q vector)
        if (ii < num_q) {
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
            }
        }
        __syncthreads();
        
        // In FoXS since c2 remains the same for all elements it is reduced to one value.
        hydration = c2 * FF_pt[num_ele];
        //if (ii == num_q && threadIdx.x == 0) {printf("Hydration is: %6.3f\n", hydration);}
        __syncthreads();
        // Calculate atomic form factor for this q
        // However to keep compatible to HyPred method we leave atom type def unchanged.
        if (ii == num_q) {
            // calculate surf_grad
            for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
                //int atomt = Ele[jj];
                //printf("B surf grads = %6.3f, %6.3f, %6.3f. \n", 
                //       surf_grad[3*jj], surf_grad[3*jj+1], surf_grad[3*jj+2]);
                /*surf_grad[3*jj]   *= hydration;
                surf_grad[3*jj+1] *= hydration;
                surf_grad[3*jj+2] *= hydration;*/
                surf_grad[3*jj]   *= c2;
                surf_grad[3*jj+1] *= c2;
                surf_grad[3*jj+2] *= c2;
                //printf("A surf grads = %6.3f, %6.3f, %6.3f. \n", 
                //       surf_grad[3*jj], surf_grad[3*jj+1], surf_grad[3*jj+2]);
            }        
        } else {
            for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
                int atomt = Ele[jj];
                if (atomt > 5) {  // Which means this is a hydrogen
                    FF_full[ii*num_atom2 + jj] = FF_pt[0];
                    FF_full[ii*num_atom2 + jj] += hydration * V[jj];
                } else {          // Heavy atoms - do the same as before
                    FF_full[ii*num_atom2 + jj] = FF_pt[atomt];
                    FF_full[ii*num_atom2 + jj] += hydration * V[jj];
                }
            }
        }
        if (threadIdx.x == 0) FF_full[ii * num_atom2 + num_atom + 1] = FF_pt[num_ele];
    }
}


__global__ void __launch_bounds__(1024,2) scat_calc (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float *sigma2,  
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *FF_full) {

    float q_pt, sigma2_pt; 

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q_S_ref_dS[ii];
        sigma2_pt = sigma2[ii];

        // Calculate scattering for Aq
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float S_calccs = 0.0;
            float f_ptxcs = 0.0;
            float f_ptycs = 0.0;
            float f_ptzcs = 0.0;
            for (int kk = 0; kk < num_atom; kk++) {
                // for every atom kk
                float FF_kj = FF_full[ii * num_atom2 + jj] * FF_full[ii *num_atom2 + kk];
                if (q_pt == 0.0 || kk == jj) {
                    S_calccs += FF_kj;
                } else {
                    /*float dx = coord[3*kk+0] - atom1x;
                    float dy = coord[3*kk+1] - atom1y;
                    float dz = coord[3*kk+2] - atom1z;*/
                    float dx = atom1x - coord[3*kk+0];
                    float dy = atom1y - coord[3*kk+1];
                    float dz = atom1z - coord[3*kk+2];
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    float qr = q_pt * r; 
                    float sqr = sin(qr) / qr;
                    float dsqr = cos(qr) - sqr;
                    float prefac = FF_kj * dsqr / r / r;
                    prefac += prefac;
                    S_calccs += FF_kj * sqr;
                    f_ptxcs += prefac * dx;
                    f_ptycs += prefac * dy;
                    f_ptzcs += prefac * dz;
                }
            }
            S_calcc[ii*num_atom2+jj] = S_calccs;
            f_ptxc[ii*num_atom2+jj] = f_ptxcs;
            f_ptyc[ii*num_atom2+jj] = f_ptycs;
            f_ptzc[ii*num_atom2+jj] = f_ptzcs;
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_atom2];
        __syncthreads();


        if (threadIdx.x == 0) {
            Aq[ii] = S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2_pt;
            Aq[ii] += Aq[ii];
        }
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
        }
    }
}

__global__ void __launch_bounds__(1024,2) scat_calc_surf_grad (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float *sigma2,  
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *surf_grad,
    float *FF_full) {

    float q_pt, sigma2_pt, FF_w; 

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q_S_ref_dS[ii];
        sigma2_pt = sigma2[ii];
        FF_w = FF_full[ii * num_atom2 + num_ele + 1]; // Water form factor at this q
        // Calculate scattering for Aq
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float S_calccs = 0.0;
            float f_ptxcs = 0.0;
            float f_ptycs = 0.0;
            float f_ptzcs = 0.0;
            float FF_j = FF_full[ii * num_atom2 + jj];
            for (int kk = 0; kk < num_atom; kk++) {
                float FF_kj = FF_j * FF_full[ii * num_atom2 + kk];
                float dx = atom1x - coord[3*kk+0];
                float dy = atom1y - coord[3*kk+1];
                float dz = atom1z - coord[3*kk+2];
                /*float dx = coord[3*kk+0] - atom1x;
                float dy = coord[3*kk+1] - atom1y;
                float dz = coord[3*kk+2] - atom1z;*/
                float r = sqrt(dx*dx+dy*dy+dz*dz);
                float prefac = 0.0;
                float sqr = 1.0;
                if (kk == jj) r = 1.0;
                if (kk == jj || q_pt == 0.0) {
                    sqr = 1.0;
                } else {
                    float qr = q_pt * r; 
                    sqr = sin(qr) / qr;
                    float dsqr = cos(qr) - sqr;
                    prefac *= sqr;
                    prefac += FF_kj * dsqr;
                }
                prefac += prefac;
                prefac = prefac / r / r;
                S_calccs += FF_kj * sqr;
                f_ptxcs += prefac * dx;
                f_ptycs += prefac * dy;
                f_ptzcs += prefac * dz;

                prefac = FF_j * sqr * FF_w;
                prefac += prefac;
                //printf("sqr = %6.3f, FF_j = %6.3f, prod = %6.3f \n", sqr, FF_j, sqr*FF_j);
                //printf("surf grads = %6.3f, %6.3f, %6.3f. \n", surf_grad[3*jj], surf_grad[3*jj+1], surf_grad[3*jj+2]);
                f_ptxcs += prefac * surf_grad[3*jj  ];
                f_ptycs += prefac * surf_grad[3*jj+1];
                f_ptzcs += prefac * surf_grad[3*jj+2];
            }
            S_calcc[ii*num_atom2+jj] = S_calccs;
            f_ptxc[ii*num_atom2+jj] = f_ptxcs;
            f_ptyc[ii*num_atom2+jj] = f_ptycs;
            f_ptzc[ii*num_atom2+jj] = f_ptzcs;
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_atom2];
        __syncthreads();


        if (threadIdx.x == 0) {
            Aq[ii] = S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2_pt;
            Aq[ii] += Aq[ii];
        }
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
        }
    }
}

__global__ void __launch_bounds__(1024,2) scat_calc_EMA (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    double *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float *sigma2,  
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *FF_full,
    double *S_old,
    double EMA_norm) {

    // EMA_norm is computed on the host. See Chen & Hub, Biophysics 2015 2573-2584.

    float q_pt, sigma2_pt; 

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q_S_ref_dS[ii];
        sigma2_pt = sigma2[ii];
        // Calculate scattering for Aq
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float S_calccs = 0.0;
            float f_ptxcs = 0.0;
            float f_ptycs = 0.0;
            float f_ptzcs = 0.0;
            for (int kk = 0; kk < num_atom; kk++) {
                // for every atom kk
                float FF_kj = FF_full[ii * num_atom2 + jj] * FF_full[ii *num_atom2 + kk];
                if (q_pt == 0.0 || kk == jj) {
                    S_calccs += FF_kj;
                } else {
                    /*float dx = coord[3*kk+0] - atom1x;
                    float dy = coord[3*kk+1] - atom1y;
                    float dz = coord[3*kk+2] - atom1z;*/
                    float dx = atom1x - coord[3*kk+0];
                    float dy = atom1y - coord[3*kk+1];
                    float dz = atom1z - coord[3*kk+2];
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    float qr = q_pt * r; 
                    float sqr = sin(qr) / qr;
                    float dsqr = cos(qr) - sqr;
                    float prefac = FF_kj * dsqr / r / r;
                    prefac += prefac;
                    S_calccs += FF_kj * sqr;
                    f_ptxcs += prefac * dx;
                    f_ptycs += prefac * dy;
                    f_ptzcs += prefac * dz;
                }
            }
            S_calcc[ii*num_atom2+jj] = S_calccs;
            f_ptxc[ii*num_atom2+jj] = f_ptxcs;
            f_ptyc[ii*num_atom2+jj] = f_ptycs;
            f_ptzc[ii*num_atom2+jj] = f_ptzcs;
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = (double) S_calcc[ii * num_atom2];
        __syncthreads();
        
        if (threadIdx.x == 0) {
            
            // Here comes in the past scat
            // Scat is calced to (S_new + ((N-1) / N) S_old) / N-1
            // Remember to convert S_new to double or set an array for it.
            S_calc[ii] += S_old[ii] * (EMA_norm - 1.0);
            S_calc[ii] /= EMA_norm;
            // Update old scattering
            S_old[ii] = S_calc[ii];
            
            Aq[ii] = (float)S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2_pt;
            Aq[ii] += Aq[ii];
        }
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
        }
    }
}

__global__ void __launch_bounds__(1024,2) scat_calc_EMA_surf_grad (
//__global__ void scat_calc_EMA_surf_grad (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    double *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float *sigma2,  
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *surf_grad,
    float *FF_full,
    double *S_old,
    double EMA_norm) {

    // EMA_norm is computed on the host. See Chen & Hub, Biophysics 2015 2573-2584.

    float q_pt, sigma2_pt; 

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q_S_ref_dS[ii];
        sigma2_pt = sigma2[ii];
        // Calculate scattering for Aq
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            // For surface gradient
            float S_calccs = 0.0;
            float f_ptxcs = 0.0;
            float f_ptycs = 0.0;
            float f_ptzcs = 0.0;
            float FF_j = FF_full[ii * num_atom2 + jj];
            for (int kk = 0; kk < num_atom; kk++) {
                float FF_kj = FF_j * FF_full[ii * num_atom2 + kk];
                /*if (kk == jj) {
                    S_calccs += FF_kj;
                } else if (q_pt == 0.0) {
                    float dx = coord[3*kk+0] - atom1x;
                    float dy = coord[3*kk+1] - atom1y;
                    float dz = coord[3*kk+2] - atom1z;
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    float prefac = surf_grad[3*jj+0] * dx; 
                    prefac += surf_grad[3*jj+1] * dy;
                    prefac += surf_grad[3*jj+2] * dz;
                    prefac *= FF_j / r / r;
                    prefac += prefac;
                    S_calccs += FF_kj;
                    f_ptxcs += prefac * dx;
                    f_ptycs += prefac * dy;
                    f_ptzcs += prefac * dz;
                } else {
                    float dx = coord[3*kk+0] - atom1x;
                    float dy = coord[3*kk+1] - atom1y;
                    float dz = coord[3*kk+2] - atom1z;
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    float qr = q_pt * r; 
                    float sqr = sin(qr) / qr;
                    float dsqr = cos(qr) - sqr;
                    float prefac = surf_grad[3*jj+0] * dx; 
                    prefac += surf_grad[3*jj+1] * dy;
                    prefac += surf_grad[3*jj+2] * dz;
                    prefac *= FF_j * sqr; 
                    prefac += FF_kj * dsqr / r / r;
                    prefac += prefac;
                    S_calccs += FF_kj * sqr;
                    f_ptxcs += prefac * dx;
                    f_ptycs += prefac * dy;
                    f_ptzcs += prefac * dz;
                }*/
                // for every atom kk
                float dx = coord[3*kk+0] - atom1x;
                float dy = coord[3*kk+1] - atom1y;
                float dz = coord[3*kk+2] - atom1z;
                /*float dx = atom1x - coord[3*kk+0];
                float dy = atom1y - coord[3*kk+1];
                float dz = atom1z - coord[3*kk+2];*/
                float r = sqrt(dx*dx+dy*dy+dz*dz);
                float prefac = surf_grad[3*jj+0] * dx; 
                prefac += surf_grad[3*jj+1] * dy;
                prefac += surf_grad[3*jj+2] * dz;
                prefac *= FF_j; 
                float sqr = 1.0;
                if (kk == jj) r = 1.0;
                if (kk == jj || q_pt == 0.0) {
                    sqr = 1.0;
                } else {
                    float qr = q_pt * r; 
                    sqr = sin(qr) / qr;
                    float dsqr = cos(qr) - sqr;
                    prefac *= sqr;
                    prefac += FF_kj * dsqr;
                }
                prefac += prefac;
                prefac = prefac / r / r;
                //printf("sqr = %6.3f, FF_kj = %6.3f, prod = %6.3f \n", sqr, FF_kj, sqr*FF_kj);
                S_calccs += FF_kj * sqr;
                f_ptxcs += prefac * dx;
                f_ptycs += prefac * dy;
                f_ptzcs += prefac * dz;
            }
            //printf("S_calccs(q=%.3f) = %6.3f\n", q_pt, S_calccs);
            S_calcc[ii*num_atom2+jj] = S_calccs;
            f_ptxc[ii*num_atom2+jj] = f_ptxcs;
            f_ptyc[ii*num_atom2+jj] = f_ptycs;
            f_ptzc[ii*num_atom2+jj] = f_ptzcs;
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = (double) S_calcc[ii * num_atom2];
        __syncthreads();
        
        if (threadIdx.x == 0) {
            
            // Here comes in the past scat
            // Scat is calced to (S_new + ((N-1) / N) S_old) / N-1
            // Remember to convert S_new to double or set an array for it.
            S_calc[ii] += S_old[ii] * (EMA_norm - 1.0);
            S_calc[ii] /= EMA_norm;
            // Update old scattering
            S_old[ii] = S_calc[ii];
            
            Aq[ii] = (float)S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2_pt;
            Aq[ii] += Aq[ii];
        }
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
        }
    }
}

__global__ void __launch_bounds__(1024,2) scat_calc_bin (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float *sigma2,
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *FF_full
    /*float *q_a_r,
    float *q_a_rx,
    float *q_a_ry,
    float *q_a_rz*/) { 

    // q_a_r is a 3D matrix of dimension num_q, 1024, 401
    // every q will use a slice of that matrix, and every atom jj % 1024 will use an array
    // of the slice, recording the FFT amplitude.

    float q_pt; 
    //__shared__ float sqr[256];  // This is binned sin(q * r) / (q * r)
    //__shared__ float csqrr[256]; // This is binned (cos(q * r) - sin(q * r) / (q * r)) / r^2
    __shared__ int q_a_r [1024];
    __shared__ int q_a_rx[1024];
    __shared__ int q_a_ry[1024];
    __shared__ int q_a_rz[1024];
    /*__shared__ float q_a_r2 [1024];
    __shared__ float q_a_rx2[1024];
    __shared__ float q_a_ry2[1024];
    __shared__ float q_a_rz2[1024];*/
 
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q_S_ref_dS[ii];
        for (int kk = threadIdx.x; kk < 1024; kk += blockDim.x) {
            q_a_r [kk] = 0;
            q_a_rx[kk] = 0;
            q_a_ry[kk] = 0;
            q_a_rz[kk] = 0;
        }
        // Determine the sqr and csqrr
        /*for (int jj = threadIdx.x; jj < 256; jj += blockDim.x) {
            float r = (float)jj * 0.5 + 0.25;
            float qr = q_pt * r;
            sqr[jj] = sin(qr) / qr;
            float dsqr = cos(qr) - sqr[jj];
            csqrr[jj] =  dsqr / r / r;
        }*/
        //__syncthreads();
        // Calculate scattering for Aq
        for (int jj = 0; jj < num_atom; jj ++) {
            /*for (int kk = threadIdx.x; kk < 1024; kk += blockDim.x) {
                q_a_r [kk] = 0;
                q_a_rx[kk] = 0;
                q_a_ry[kk] = 0;
                q_a_rz[kk] = 0;*/
                /*q_a_r2 [kk] = 0.0;
                q_a_rx2[kk] = 0.0;
                q_a_ry2[kk] = 0.0;
                q_a_rz2[kk] = 0.0;*/
            //}

            //__syncthreads(); // This one is essential
            // for every atom jj
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float atom1FF = FF_full[ii*num_atom2 +jj];
            //__syncthreads(); // This one is essential
            for (int kk = threadIdx.x; kk < num_atom; kk+= blockDim.x) {
                // for every atom kk
                //float FF_kj = atom1FF * FF_full[ii *num_atom2 + kk];
                float FF_kj = FF_full[ii *num_atom2 + kk];
                //if (q_pt == 0.0 || kk == jj) {
                //    S_calccs += FF_kj;
                //} else {
                float dx = coord[3*kk+0] - atom1x;
                float dy = coord[3*kk+1] - atom1y;
                float dz = coord[3*kk+2] - atom1z;
                float r = sqrt(dx*dx+dy*dy+dz*dz);
                //if (r < 64) {
                    int idz = r += r; // Because we're using 0.5 A bins.
                    /*int ida = threadIdx.x / 256 * 256;
                    atomicAdd(&q_a_r [idz+ida], (int)(FF_kj * 1e6));
                    atomicAdd(&q_a_rx[idz+ida], (int)(2e6 * FF_kj * dx));
                    atomicAdd(&q_a_ry[idz+ida], (int)(2e6 * FF_kj * dy));
                    atomicAdd(&q_a_rz[idz+ida], (int)(2e6 * FF_kj * dz));*/
                    int dumm = atom1FF * FF_kj * 1e4;
                    atomicAdd(&q_a_r [idz], dumm);
                    dumm = atom1FF * FF_kj * dx * 2e4;
                    atomicAdd(&q_a_rx[idz], dumm);
                    dumm = atom1FF * FF_kj * dy * 2e4;
                    atomicAdd(&q_a_ry[idz], dumm);
                    dumm = atom1FF * FF_kj * dz * 2e4;
                    atomicAdd(&q_a_rz[idz], dumm);

                    /*atomicAdd(&q_a_r [idz], (int)(FF_kj * 1e4));
                    atomicAdd(&q_a_rx[idz], (int)(2e4 * FF_kj * dx));
                    atomicAdd(&q_a_ry[idz], (int)(2e4 * FF_kj * dy));
                    atomicAdd(&q_a_rz[idz], (int)(2e4 * FF_kj * dz));*/
                        /*float qr = q_pt * r; 
                        float sqr = sin(qr) / qr;
                        float dsqr = cos(qr) - sqr;
                        float prefac = FF_kj * dsqr / r / r;
                        prefac += prefac;
                        S_calccs += FF_kj * sqr;
                        f_ptxcs += prefac * dx;
                        f_ptycs += prefac * dy;
                        f_ptzcs += prefac * dz;*/
                    //}
                //}
            }

            /*for (int stride = 512; stride > 128; stride >>= 1) {
                __syncthreads();
                for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                    q_a_r[iAccum] += q_a_r[iAccum + stride];
                    q_a_rx[iAccum] += q_a_rx[iAccum + stride];
                    q_a_ry[iAccum] += q_a_ry[iAccum + stride];
                    q_a_rz[iAccum] += q_a_rz[iAccum + stride];
                }
            }*/
            __syncthreads();

            for (int kk = threadIdx.x; kk < 256; kk += blockDim.x) {
                float r = (float)kk * 0.5 + 0.25;
                float qr = q_pt * r;
                float sqr = sin(qr) / qr;
                float dsqr = cos(qr) - sqr;
                float csqrr = dsqr / r / r;
                //atomicAdd(&S_calcc[ii*num_atom2+jj],(float)( 1e-6 * q_a_r[kk]  *   sqr[kk]));
                //atomicAdd(&f_ptxc[ii*num_atom2+jj], (float)( 1e-6 * q_a_rx[kk] * csqrr[kk]));
                //atomicAdd(&f_ptyc[ii*num_atom2+jj], (float)( 1e-6 * q_a_ry[kk] * csqrr[kk]));
                //atomicAdd(&f_ptzc[ii*num_atom2+jj], (float)( 1e-6 * q_a_rz[kk] * csqrr[kk]));
                /*atomicAdd(&S_calcc[ii*num_atom2+jj],(float)( 1e-6 * q_a_r[kk]  *   sqr));
                atomicAdd(&f_ptxc[ii*num_atom2+jj], (float)( 1e-6 * q_a_rx[kk] * csqrr));
                atomicAdd(&f_ptyc[ii*num_atom2+jj], (float)( 1e-6 * q_a_ry[kk] * csqrr));
                atomicAdd(&f_ptzc[ii*num_atom2+jj], (float)( 1e-6 * q_a_rz[kk] * csqrr));*/
                /*q_a_r2[kk] = q_a_r[kk] * sqr;
                q_a_rx2[kk] = q_a_rx[kk] * csqrr;
                q_a_ry2[kk] = q_a_ry[kk] * csqrr;
                q_a_rz2[kk] = q_a_rz[kk] * csqrr;*/
                atomicAdd(&S_calcc[ii*num_atom2+jj],q_a_r[kk]  *   sqr);
                q_a_r[kk] = 0;
                atomicAdd(&f_ptxc[ii*num_atom2+jj], q_a_rx[kk] * csqrr);
                q_a_rx[kk] = 0;
                atomicAdd(&f_ptyc[ii*num_atom2+jj], q_a_ry[kk] * csqrr);
                q_a_ry[kk] = 0;
                atomicAdd(&f_ptzc[ii*num_atom2+jj], q_a_rz[kk] * csqrr);
                q_a_rz[kk] = 0;
            }
            __syncthreads();

            /*
            for (int stride = 1024 / 2; stride > 0; stride >>= 1) {
                __syncthreads();
                for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                    q_a_r2[iAccum] += q_a_r2[stride + iAccum];
                    q_a_rx2[iAccum] += q_a_rx2[stride + iAccum];
                    q_a_ry2[iAccum] += q_a_ry2[stride + iAccum];
                    q_a_rz2[iAccum] += q_a_rz2[stride + iAccum];
                }
            }
            //__syncthreads();
            if (threadIdx.x == 0) {
                S_calcc[ii*num_atom2+jj]+= q_a_r2[0] ;
                f_ptxc [ii*num_atom2+jj]+= q_a_rx2[0];
                f_ptyc [ii*num_atom2+jj]+= q_a_ry2[0];
                f_ptzc [ii*num_atom2+jj]+= q_a_rz2[0];
            }
            */
 
        }
        
        //if (ii == 0 && threadIdx.x == 0) printf("\nS_calcc[0] = %.3f \n",S_calcc[0]);

        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        //if (threadIdx.x == 0) {
            S_calc[ii] = S_calcc[ii * num_atom2]*1e-4;
            Aq[ii] = S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2[ii];
            Aq[ii] += Aq[ii];
        //}
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
        }
    }
}
__global__ void __launch_bounds__(1024,2) scat_calc_bin_unroll (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float *sigma2,
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *FF_full   
    ) { 

    float q_pt; 
    /*__shared__ int q_a_r [4][511];
    __shared__ int q_a_rx[4][511];
    __shared__ int q_a_ry[4][511];
    __shared__ int q_a_rz[4][511];*/
    /*__shared__ int q_a_r [1024];
    __shared__ int q_a_rx[1024];
    __shared__ int q_a_ry[1024];
    __shared__ int q_a_rz[1024];
    __shared__ int q_a_r2 [1024];
    __shared__ int q_a_rx2[1024];
    __shared__ int q_a_ry2[1024];
    __shared__ int q_a_rz2[1024];*/
    __shared__ int q_a_r [512];
    __shared__ int q_a_rx[512];
    __shared__ int q_a_ry[512];
    __shared__ int q_a_rz[512];
    __shared__ int q_a_r2 [512];
    __shared__ int q_a_rx2[512];
    __shared__ int q_a_ry2[512];
    __shared__ int q_a_rz2[512];
    /*__shared__ float atomx[4];
    __shared__ float atomy[4];
    __shared__ float atomz[4];
    __shared__ float atomFF[4];*/
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q_S_ref_dS[ii];
        for (int kk = threadIdx.x; kk < 256; kk += blockDim.x) {
            /*for (int ll = 0; ll < 4; ll++) {
                q_a_r[ll][kk] = 0;
                q_a_rx[ll][kk] = 0;
                q_a_ry[ll][kk] = 0;
                q_a_rz[ll][kk] = 0;
            }*/
            q_a_r [kk] = 0;
            q_a_rx[kk] = 0;
            q_a_ry[kk] = 0;
            q_a_rz[kk] = 0;
            q_a_r2 [kk] = 0;
            q_a_rx2[kk] = 0;
            q_a_ry2[kk] = 0;
            q_a_rz2[kk] = 0;
        }
        for (int jj = 0; jj < num_atom; jj += 2) {
            // for every atom jj
            /*for (int kk = 0; kk < 4; kk++) {
                atomx[kk] = coord[3*jj+3*kk+0];
                atomy[kk] = coord[3*jj+3*kk+1];
                atomz[kk] = coord[3*jj+3*kk+2];
                atomFF[kk] = FF_full[ii*num_atom2 + jj + 3 * kk];
            }*/
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float atom2x = coord[3*jj+3];
            float atom2y = coord[3*jj+4];
            float atom2z = coord[3*jj+5];
            float atom1FF = FF_full[ii*num_atom2 +jj];
            float atom2FF = FF_full[ii*num_atom2 + jj + 1];
            for (int kk = threadIdx.x; kk < num_atom; kk+= blockDim.x) {
                // for every atom kk
                float FF_kj = FF_full[ii *num_atom2 + kk];
                /*for (int ll = 0; ll < 4; ll++) {

                    float dx = coord[3*kk+0] - atomx[ll];
                    float dy = coord[3*kk+1] - atomy[ll];
                    float dz = coord[3*kk+2] - atomz[ll];
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    int idz = r += r; // Because we're using 0.5 A bins.
                    int dumm = atomFF[ll] * FF_kj * 1e4;
                    atomicAdd(&q_a_r [ll][idz], dumm);
                    dumm = atomFF[ll] * FF_kj * dx * 2e4;
                    atomicAdd(&q_a_rx[ll][idz], dumm);
                    dumm = atomFF[ll] * FF_kj * dy * 2e4;
                    atomicAdd(&q_a_ry[ll][idz], dumm);
                    dumm = atomFF[ll] * FF_kj * dz * 2e4;
                    atomicAdd(&q_a_rz[ll][idz], dumm);

                }*/
                float x = coord[3*kk+0];
                float y = coord[3*kk+1];
                float z = coord[3*kk+2];
                float dx = x - atom1x;
                float dy = y - atom1y;
                float dz = z - atom1z;
                /*float dx = coord[3*kk+0] - atom1x;
                float dy = coord[3*kk+1] - atom1y;
                float dz = coord[3*kk+2] - atom1z;*/
                float r = sqrt(dx*dx+dy*dy+dz*dz);
                int idz = r += r; // Because we're using 0.5 A bins.
                //if (idz < 256) {
                int dumm = atom1FF * FF_kj * 10000;
                atomicAdd(&q_a_r [idz], dumm);
                dumm = atom1FF * FF_kj * dx * 20000;
                atomicAdd(&q_a_rx[idz], dumm);
                dumm = atom1FF * FF_kj * dy * 20000;
                atomicAdd(&q_a_ry[idz], dumm);
                dumm = atom1FF * FF_kj * dz * 20000;
                atomicAdd(&q_a_rz[idz], dumm);
/*
                // Try to avoid double mul
                int dumm = atom1FF * 1e4;
                dumm *= FF_kj;
                atomicAdd(&q_a_r [idz], dumm);
                dumm = dx * 2e4;
                dumm *= atom1FF * FF_kj;
                atomicAdd(&q_a_rx[idz], dumm);
                dumm = dy * 2e4;
                dumm *= atom1FF * FF_kj;
                atomicAdd(&q_a_ry[idz], dumm);
                dumm = dz * 2e4;
                dumm *= atom1FF * FF_kj;
                atomicAdd(&q_a_rz[idz], dumm);
*/
                //}
                /*dx = coord[3*kk+0] - atom2x;
                dy = coord[3*kk+1] - atom2y;
                dz = coord[3*kk+2] - atom2z;*/
                dx = x - atom2x;
                dy = y - atom2y;
                dz = z - atom2z;
                r = sqrt(dx*dx+dy*dy+dz*dz);
                idz = r += r; // Because we're using 0.5 A bins.
                //if (idz < 256) {
                //int dumm = atom2FF * FF_kj * 1e4;
                dumm = atom2FF * FF_kj * 10000;
                atomicAdd(&q_a_r2 [idz], dumm);
                dumm = atom2FF * FF_kj * dx * 20000;
                atomicAdd(&q_a_rx2[idz], dumm);
                dumm = atom2FF * FF_kj * dy * 20000;
                atomicAdd(&q_a_ry2[idz], dumm);
                dumm = atom2FF * FF_kj * dz * 20000;
                atomicAdd(&q_a_rz2[idz], dumm);

                //}
/*                dumm = atom1FF * 1e4;
                dumm *= FF_kj;
                atomicAdd(&q_a_r2 [idz], dumm);
                dumm = dx * 2e4;
                dumm *= atom1FF * FF_kj;
                atomicAdd(&q_a_rx2[idz], dumm);
                dumm = dy * 2e4;
                dumm *= atom1FF * FF_kj;
                atomicAdd(&q_a_ry2[idz], dumm);
                dumm = dz * 2e4;
                dumm *= atom1FF * FF_kj;
                atomicAdd(&q_a_rz2[idz], dumm);*/
            }

            __syncthreads();

            for (int kk = threadIdx.x; kk < 256; kk += blockDim.x) {
                float r = (float)kk * 0.5 + 0.25;
                float qr = q_pt * r;
                float sqr = sin(qr) / qr;
                float dsqr = cos(qr) - sqr;
                float csqrr = dsqr / r / r;
                /*for (int ll = 0; ll < 4; ll++) {
                    atomicAdd(&S_calcc[ii*num_atom2+jj],q_a_r[ll][kk]  *   sqr);
                    q_a_r[ll][kk] = 0;
                    atomicAdd(&f_ptxc[ii*num_atom2+jj],q_a_rx[ll][kk]  *   csqrr);
                    q_a_rx[ll][kk] = 0;
                    atomicAdd(&f_ptyc[ii*num_atom2+jj],q_a_ry[ll][kk]  *   csqrr);
                    q_a_ry[ll][kk] = 0;
                    atomicAdd(&f_ptzc[ii*num_atom2+jj],q_a_rz[ll][kk]  *   csqrr);
                    q_a_rz[ll][kk] = 0;
                }*/
                atomicAdd(&S_calcc[ii*num_atom2+jj],q_a_r[kk]  *   sqr);
                atomicAdd(&S_calcc[ii*num_atom2+jj+1],q_a_r2[kk]  *   sqr);
                q_a_r[kk] = 0;
                q_a_r2[kk] = 0;
                atomicAdd(&f_ptxc[ii*num_atom2+jj], q_a_rx[kk] * csqrr);
                atomicAdd(&f_ptxc[ii*num_atom2+jj+1], q_a_rx2[kk] * csqrr);
                q_a_rx[kk] = 0;
                q_a_rx2[kk] = 0;
                atomicAdd(&f_ptyc[ii*num_atom2+jj], q_a_ry[kk] * csqrr);
                atomicAdd(&f_ptyc[ii*num_atom2+jj+1], q_a_ry2[kk] * csqrr);
                q_a_ry[kk] = 0;
                q_a_ry2[kk] = 0;
                atomicAdd(&f_ptzc[ii*num_atom2+jj], q_a_rz[kk] * csqrr);
                atomicAdd(&f_ptzc[ii*num_atom2+jj+1], q_a_rz2[kk] * csqrr);
                q_a_rz[kk] = 0;
                q_a_rz2[kk] = 0;
            }
            __syncthreads();

            /*
            for (int stride = 1024 / 2; stride > 0; stride >>= 1) {
                __syncthreads();
                for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                    q_a_r2[iAccum] += q_a_r2[stride + iAccum];
                    q_a_rx2[iAccum] += q_a_rx2[stride + iAccum];
                    q_a_ry2[iAccum] += q_a_ry2[stride + iAccum];
                    q_a_rz2[iAccum] += q_a_rz2[stride + iAccum];
                }
            }
            //__syncthreads();
            if (threadIdx.x == 0) {
                S_calcc[ii*num_atom2+jj]+= q_a_r2[0] ;
                f_ptxc [ii*num_atom2+jj]+= q_a_rx2[0];
                f_ptyc [ii*num_atom2+jj]+= q_a_ry2[0];
                f_ptzc [ii*num_atom2+jj]+= q_a_rz2[0];
            }
            */
 
        }
        
        //if (ii == 0 && threadIdx.x == 0) printf("\nS_calcc[0] = %.3f \n",S_calcc[0]);

        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        //if (threadIdx.x == 0) {
            S_calc[ii] = S_calcc[ii * num_atom2]*1e-4;
            Aq[ii] = S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2[ii];
            Aq[ii] += Aq[ii];
        //}
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
        }
    }
}
__global__ void __launch_bounds__(1024,2) scat_calc_bin_unroll2 (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float *sigma2,
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *FF_full   
    ) { 

    float q_pt; 
    __shared__ int q_a_r [2048];
    __shared__ int q_a_rx[2048];
    __shared__ int q_a_ry[2048];
    __shared__ int q_a_rz[2048];
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q_S_ref_dS[ii];
        for (int kk = threadIdx.x; kk < 2048; kk += blockDim.x) {
            q_a_r[kk] = 0;
            q_a_rx[kk] = 0;
            q_a_ry[kk] = 0;
            q_a_rz[kk] = 0;
        }
        for (int jj = 0; jj < num_atom; jj += 3) {
            // for every atom jj
            /*for (int kk = 0; kk < 4; kk++) {
                atomx[kk] = coord[3*jj+3*kk+0];
                atomy[kk] = coord[3*jj+3*kk+1];
                atomz[kk] = coord[3*jj+3*kk+2];
                atomFF[kk] = FF_full[ii*num_atom2 + jj + 3 * kk];
            }*/
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float atom2x = coord[3*jj+3];
            float atom2y = coord[3*jj+4];
            float atom2z = coord[3*jj+5];
            float atom3x = coord[3*jj+6];
            float atom3y = coord[3*jj+7];
            float atom3z = coord[3*jj+8];
            //float atom4x = coord[3*jj+9];
            //float atom4y = coord[3*jj+10];
            //float atom4z = coord[3*jj+11];
            float atom1FF = FF_full[ii*num_atom2 + jj];
            float atom2FF = FF_full[ii*num_atom2 + jj + 1];
            float atom3FF = FF_full[ii*num_atom2 + jj + 2];
            //float atom4FF = FF_full[ii*num_atom2 + jj + 3];
            for (int kk = threadIdx.x; kk < num_atom; kk+= blockDim.x) {
                // for every atom kk
                float FF_kj = FF_full[ii *num_atom2 + kk];
                float x = coord[3*kk+0];
                float y = coord[3*kk+1];
                float z = coord[3*kk+2];
                float dx = x - atom1x;
                float dy = y - atom1y;
                float dz = z - atom1z;
                float r = sqrt(dx*dx+dy*dy+dz*dz);
                int idz = r += r; // Because we're using 0.5 A bins.
                int dumm = atom1FF * FF_kj * 10000;
                atomicAdd(&q_a_r [idz], dumm);
                dumm = atom1FF * FF_kj * dx * 20000;
                atomicAdd(&q_a_rx[idz], dumm);
                dumm = atom1FF * FF_kj * dy * 20000;
                atomicAdd(&q_a_ry[idz], dumm);
                dumm = atom1FF * FF_kj * dz * 20000;
                atomicAdd(&q_a_rz[idz], dumm);

                dx = x - atom2x;
                dy = y - atom2y;
                dz = z - atom2z;
                r = sqrt(dx*dx+dy*dy+dz*dz);
                idz = r += r; // Because we're using 0.5 A bins.
                dumm = atom2FF * FF_kj * 10000;
                atomicAdd(&q_a_r [idz+512], dumm);
                dumm = atom2FF * FF_kj * dx * 20000;
                atomicAdd(&q_a_rx[idz+512], dumm);
                dumm = atom2FF * FF_kj * dy * 20000;
                atomicAdd(&q_a_ry[idz+512], dumm);
                dumm = atom2FF * FF_kj * dz * 20000;
                atomicAdd(&q_a_rz[idz+512], dumm);

                dx = x - atom3x;
                dy = y - atom3y;
                dz = z - atom3z;
                r = sqrt(dx*dx+dy*dy+dz*dz);
                idz = r += r; // Because we're using 0.5 A bins.
                dumm = atom3FF * FF_kj * 10000;
                atomicAdd(&q_a_r [idz+1024], dumm);
                dumm = atom3FF * FF_kj * dx * 20000;
                atomicAdd(&q_a_rx[idz+1024], dumm);
                dumm = atom3FF * FF_kj * dy * 20000;
                atomicAdd(&q_a_ry[idz+1024], dumm);
                dumm = atom3FF * FF_kj * dz * 20000;
                atomicAdd(&q_a_rz[idz+1024], dumm);

/*                dx = x - atom4x;
                dy = y - atom4y;
                dz = z - atom4z;
                r = sqrt(dx*dx+dy*dy+dz*dz);
                idz = r += r; // Because we're using 0.5 A bins.
                dumm = atom4FF * FF_kj * 10000;
                atomicAdd(&q_a_r [idz+1536], dumm);
                dumm = atom4FF * FF_kj * dx * 20000;
                atomicAdd(&q_a_rx[idz+1536], dumm);
                dumm = atom4FF * FF_kj * dy * 20000;
                atomicAdd(&q_a_ry[idz+1536], dumm);
                dumm = atom4FF * FF_kj * dz * 20000;
                atomicAdd(&q_a_rz[idz+1536], dumm);
*/
            }

            __syncthreads();

            for (int kk = threadIdx.x; kk < 1536; kk += blockDim.x) {
                float r = (float)(kk % 512 * 0.5 + 0.25);
                float qr = q_pt * r;
                float sqr = sin(qr) / qr;
                float dsqr = cos(qr) - sqr;
                float csqrr = dsqr / r / r;
                    atomicAdd(&S_calcc[ii*num_atom2+jj],q_a_r[kk]  *   sqr);
                    q_a_r[kk] = 0;
                    atomicAdd(&f_ptxc[ii*num_atom2+jj],q_a_rx[kk]  *   csqrr);
                    q_a_rx[kk] = 0;
                    atomicAdd(&f_ptyc[ii*num_atom2+jj],q_a_ry[kk]  *   csqrr);
                    q_a_ry[kk] = 0;
                    atomicAdd(&f_ptzc[ii*num_atom2+jj],q_a_rz[kk]  *   csqrr);
                    q_a_rz[kk] = 0;
            }
            __syncthreads();

            /*
            for (int stride = 1024 / 2; stride > 0; stride >>= 1) {
                __syncthreads();
                for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                    q_a_r2[iAccum] += q_a_r2[stride + iAccum];
                    q_a_rx2[iAccum] += q_a_rx2[stride + iAccum];
                    q_a_ry2[iAccum] += q_a_ry2[stride + iAccum];
                    q_a_rz2[iAccum] += q_a_rz2[stride + iAccum];
                }
            }
            //__syncthreads();
            if (threadIdx.x == 0) {
                S_calcc[ii*num_atom2+jj]+= q_a_r2[0] ;
                f_ptxc [ii*num_atom2+jj]+= q_a_rx2[0];
                f_ptyc [ii*num_atom2+jj]+= q_a_ry2[0];
                f_ptzc [ii*num_atom2+jj]+= q_a_rz2[0];
            }
            */
 
        }
        
        //if (ii == 0 && threadIdx.x == 0) printf("\nS_calcc[0] = %.3f \n",S_calcc[0]);

        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        //if (threadIdx.x == 0) {
            S_calc[ii] = S_calcc[ii * num_atom2]*1e-4;
            Aq[ii] = S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2[ii];
            Aq[ii] += Aq[ii];
        //}
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha * 1e-4;
        }
    }
}
/*
__global__ void FF_calc_T (
    float *FF_full,
    float *FF_full_T,
    int num_q2, 
    int num_atom2) {


    int TILE_DIM = 32;
    int BLOCK_ROWS = num_atom2 * num_q2 / TILE_DIM;
    __shared__ float tile[TILE_DIM * TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = FF_full[(y+j)*width + x];

    __syncthreads();
    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        FF_full_T[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];  

}
*/

__global__ void __launch_bounds__(1024,2) scat_calc_bin_T (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    float *S_calc, 
    int num_atom,   
    int num_q,    
    int num_q2, 
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float sigma2,
    float *f_ptxc_T, 
    float *f_ptyc_T, 
    float *f_ptzc_T, 
    float *S_calcc_T, 
    int num_atom2,
    float *FF_full_T,
    float *a_r_q,
    float *a_r_qx,
    float *a_r_qy,
    float *a_r_qz) { 

    // a_r_q is a 3D matrix of dimension 896 (16*56 for now), 416, num_q2
    // every atom % 1024 will use a slice of that matrix, and every q vector will use an array
    // of the slice, recording the FFT amplitude.

    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        // Flush a_r_q

        for (int jj = threadIdx.x; jj < num_q; jj += blockDim.x) {
            for (int kk = 0; kk < 416; kk ++) {
                a_r_q [ii % 896 * num_q2 * 416 + kk * num_q2 + jj] = 0.0;
                a_r_qx[ii % 896 * num_q2 * 416 + kk * num_q2 + jj] = 0.0;
                a_r_qy[ii % 896 * num_q2 * 416 + kk * num_q2 + jj] = 0.0;
                a_r_qz[ii % 896 * num_q2 * 416 + kk * num_q2 + jj] = 0.0;
            }


        }
 
        float atom1x = coord[ii+0];
        float atom1y = coord[ii+1*num_atom];
        float atom1z = coord[ii+2*num_atom];
        __syncthreads();
        // Calculate scattering for Aq
        for (int jj = threadIdx.x; jj < num_q; jj += blockDim.x) {
            float atom1FF = FF_full_T[ii * num_q2 + jj];
            //if (ii == 0 && jj == 0) printf("\natom1FF is %.3f\n", atom1FF);
            float q_pt = q_S_ref_dS[jj];
            // for every q jj
            for (int kk = 0; kk < num_atom; kk++) {
                // for every atom kk
                float FF_kj = atom1FF * FF_full_T[kk * num_q2 + jj];
                float dx = coord[kk+0] - atom1x;
                float dy = coord[kk+1*num_atom] - atom1y;
                float dz = coord[kk+2*num_atom] - atom1z;
                float r = sqrt(dx*dx+dy*dy+dz*dz);
                /*if (ii == 0 && jj == 0 && kk == 1) {
                    printf("\natom1 = %8.3f, %8.3f, %8.3f \natom2 = %8.3f, %8.3f, %.3f, \nr01 = %.3f\n",atom1x, atom1y, atom1z, coord[kk], coord[kk+num_atom], coord[kk+2*num_atom], r);
                }*/
                if (r < 200.0) {
                    int idz = r * 2; // Because we're using 0.5 A bins.
                    /*int ida = threadIdx.x / 256 * 256;
                    atomicAdd(&q_a_r [idz+ida], (int)(FF_kj * 1e6));
                    atomicAdd(&q_a_rx[idz+ida], (int)(2e6 * FF_kj * dx));
                    atomicAdd(&q_a_ry[idz+ida], (int)(2e6 * FF_kj * dy));
                    atomicAdd(&q_a_rz[idz+ida], (int)(2e6 * FF_kj * dz));*/
                    a_r_q [ii % 896*num_q2*416+idz*num_q2+jj] += FF_kj;
                    a_r_qx[ii % 896*num_q2*416+idz*num_q2+jj] += FF_kj * dx;
                    a_r_qx[ii % 896*num_q2*416+idz*num_q2+jj] += FF_kj * dx;
                    a_r_qy[ii % 896*num_q2*416+idz*num_q2+jj] += FF_kj * dy;
                    a_r_qy[ii % 896*num_q2*416+idz*num_q2+jj] += FF_kj * dy;
                    a_r_qz[ii % 896*num_q2*416+idz*num_q2+jj] += FF_kj * dz;
                    a_r_qz[ii % 896*num_q2*416+idz*num_q2+jj] += FF_kj * dz;
    
                        /*float qr = q_pt * r; 
                        float sqr = sin(qr) / qr;
                        float dsqr = cos(qr) - sqr;
                        float prefac = FF_kj * dsqr / r / r;
                        prefac += prefac;
                        S_calccs += FF_kj * sqr;
                        f_ptxcs += prefac * dx;
                        f_ptycs += prefac * dy;
                        f_ptzcs += prefac * dz;*/
                    //}
                }    
            }
        
            /*for (int stride = 512; stride > 128; stride >>= 1) {
                __syncthreads();
                for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                    q_a_r[iAccum] += q_a_r[iAccum + stride];
                    q_a_rx[iAccum] += q_a_rx[iAccum + stride];
                    q_a_ry[iAccum] += q_a_ry[iAccum + stride];
                    q_a_rz[iAccum] += q_a_rz[iAccum + stride];
                }
            }*/
            //__syncthreads();

            float S_calccs = 0.0;
            float f_ptxcs = 0.0;
            float f_ptycs = 0.0;
            float f_ptzcs = 0.0;

            for (int kk = 0; kk < 416; kk ++) {
                float r = (float) kk * 0.5 + 0.25;
                float qr = q_pt * r;
                float sqr = sin(qr) / qr;
                float dsqr = cos(qr) - sqr;
                float csqrr = dsqr / r / r;
                //atomicAdd(&S_calcc[ii*num_atom2+jj],(float)( 1e-6 * q_a_r[kk]  *   sqr[kk]));
                //atomicAdd(&f_ptxc[ii*num_atom2+jj], (float)( 1e-6 * q_a_rx[kk] * csqrr[kk]));
                //atomicAdd(&f_ptyc[ii*num_atom2+jj], (float)( 1e-6 * q_a_ry[kk] * csqrr[kk]));
                //atomicAdd(&f_ptzc[ii*num_atom2+jj], (float)( 1e-6 * q_a_rz[kk] * csqrr[kk]));
                S_calccs += a_r_q [(ii % 896)*num_q2*416+kk*num_q2+jj] *   sqr;
                f_ptxcs  += a_r_qx[(ii % 896)*num_q2*416+kk*num_q2+jj] * csqrr;
                f_ptycs  += a_r_qy[(ii % 896)*num_q2*416+kk*num_q2+jj] * csqrr;
                f_ptzcs  += a_r_qz[(ii % 896)*num_q2*416+kk*num_q2+jj] * csqrr;
            }

            S_calcc_T[ii*num_q2+jj]+= S_calccs;
            f_ptxc_T[ii*num_q2+jj] += f_ptxcs ;
            f_ptyc_T[ii*num_q2+jj] += f_ptycs ;
            f_ptzc_T[ii*num_q2+jj] += f_ptzcs ;

        }
        
    }
    //if (blockIdx.x == 0 && threadIdx.x == 0) printf("\nS_calcc[0] = %.3f \n",S_calcc[0]);
}


__global__ void __launch_bounds__(1024,2) scat_calc_bin2 (
    float *coord, 
    int *Ele,
    float *q_S_ref_dS, 
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *Aq, 
    float alpha,   
    float k_chi,    
    float sigma2,
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    float *S_calcc, 
    int num_atom2,
    float *FF_full,
    int *q_a_r,
    int num_atom_3 // should be num_atom round up to 3072
    /*float *q_a_rx,
    float *q_a_ry,
    float *q_a_rz*/) { 

    // q_a_r is a 3D matrix of dimension 15 (1x SM), 64 (buffer), 192 (r_bin)


    extern __shared__ int buffer[];
    int *hist = buffer; // Histogram and the indices
    int size_buf = 12288; // For Kepler, use full shared memory
    float *dxyzFF = (float *)buffer; // dx, dy, dz, FF of that partition of atoms


    for (int jj = blockIdx.x; jj < num_atom; jj += gridDim.x) {
        // For every atom j
        int idx = jj % 15;
        float atom1x = coord[3*jj+0];
        float atom1y = coord[3*jj+1];
        float atom1z = coord[3*jj+2];
        int this_round = 0;
        for (int kk = this_round * 3 * blockDim.x + threadIdx.x; kk < num_atom_3; kk += 3 * blockDim.x) {
            // Flush and initialize the histogram
            for (int ll = threadIdx.x; ll < size_buf; ll += blockDim.x) hist[ll] = -2;
            __syncthreads();
            if (threadIdx.x < 192) hist[threadIdx.x] = 1;
            __syncthreads();
            
            int idy = kk;
            for (int ll = 0; ll < 3; ll ++) {
                if (idy < num_atom) {
                    float dx = coord[3*idy+0] - atom1x;
                    float dy = coord[3*idy+1] - atom1y;
                    float dz = coord[3*idy+2] - atom1z;
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    int idz = r * 2.0;
                    if (idz < 192) {
                        int hist_idx = atomicAdd(&hist[idz],1);
                        if (hist_idx < 63) {
                            hist[hist_idx * 192 + idz] = idy % 3072;
                        }
                    }
                }

                idy += blockDim.x;
            }


            __syncthreads(); 
            // Now the histogram is done, copy it to global memory
            
            for (int ll = threadIdx.x; ll < size_buf; ll += blockDim.x) {
                q_a_r[idx*64*192 + ll] = hist[ll];
                hist[ll] = 0;
            }
            __syncthreads();

            // load with coord
            for (int ll = threadIdx.x; ll < size_buf / 4; ll += blockDim.x) {
                if (3072 * this_round + ll < num_atom) {
                    dxyzFF[ll + 3 * blockDim.x] = coord[3*(3072*this_round+ll)+0] - atom1x;
                    dxyzFF[ll + 6 * blockDim.x] = coord[3*(3072*this_round+ll)+1] - atom1y; 
                    dxyzFF[ll + 9 * blockDim.x] = coord[3*(3072*this_round+ll)+2] - atom1z;
                    // We'll load FF later
                }
            }
            __syncthreads();
            for (int ii = 0; ii < num_q; ii++) {
                float atom1FF = FF_full[ii * num_atom2 + jj];
                for (int ll = threadIdx.x; ll < size_buf / 4; ll += blockDim.x) {
                    if (3072 * this_round + ll < num_atom) 
                        dxyzFF[ll] = FF_full[ii * num_atom2 + 3072 * this_round + ll];
                }
                __syncthreads();
                // Now we start to calculate the scattering
                float q_pt = q_S_ref_dS[ii];
                float dx = 0.0;
                float dy = 0.0;
                float dz = 0.0;
                float FF = 0.0;
                if (threadIdx.x < 960) {
                    for (int ll = threadIdx.x; ll < size_buf - 192; ll += 960) {
                        if (q_a_r[idx * 64 * 192 + ll + 192] >= 0) {
                            int idy = q_a_r[idx * 64 * 192 + ll + 192];
                            dx += dxyzFF[idy] * atom1FF * dxyzFF[idy + 3 * 960];
                            dy += dxyzFF[idy] * atom1FF * dxyzFF[idy + 6 * 960];
                            dz += dxyzFF[idy] * atom1FF * dxyzFF[idy + 9 * 960];
                            FF += dxyzFF[idy] * atom1FF;
                        }
                    }
                    float r = (float)(idy % 192) * 0.5 + 0.25;
                    float qr = q_pt * r;
                    float sqr = sin(qr) / qr;
                    float dsqr = cos(qr) - sqr;
                    float csqrr = dsqr / r / r;
                    atomicAdd(&S_calcc[ii * num_atom2 + jj], FF * sqr);
                    atomicAdd(&f_ptxc [ii * num_atom2 + jj], dx * csqrr);
                    atomicAdd(&f_ptyc [ii * num_atom2 + jj], dy * csqrr);
                    atomicAdd(&f_ptzc [ii * num_atom2 + jj], dz * csqrr);

                }
                __syncthreads();

            }
           

        }
    this_round++;
    }
}

__global__ void sum_S_calc (
    float *S_calcc,
    float *f_ptxc,
    float *f_ptyc,
    float *f_ptzc,
    float *S_calc,
    float *Aq,
    float *q_S_ref_dS,
    int num_q,
    int num_atom,
    int num_atom2,
    float alpha,
    float k_chi,
    float *sigma2) {

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_atom2];
        __syncthreads();
        if (threadIdx.x == 0) {
            Aq[ii] = S_calc[ii] - q_S_ref_dS[ii+num_q];
            Aq[ii] *= -alpha;
            Aq[ii] += q_S_ref_dS[ii + 2*num_q];
            Aq[ii] *= k_chi / sigma2[ii];
            Aq[ii] += Aq[ii];
        }
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * alpha;
        }
    }
}    

__global__ void force_calc (
    float *Force,
    int num_atom, 
    int num_q, 
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    int num_atom2, 
    int num_q2, 
    int *Ele,
    float force_ramp) {
    // Do column tree sum of f_ptxc for f_ptx for every atom, then assign threadIdx.x == 0 (3 * num_atoms) to Force. Force is num_atom * 3. 
    if (blockIdx.x >= num_atom) return;
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        for (int stride = num_q2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                f_ptxc[ii + iAccum * num_atom2] += f_ptxc[ii + iAccum * num_atom2 + stride * num_atom2];
                f_ptyc[ii + iAccum * num_atom2] += f_ptyc[ii + iAccum * num_atom2 + stride * num_atom2];
                f_ptzc[ii + iAccum * num_atom2] += f_ptzc[ii + iAccum * num_atom2 + stride * num_atom2];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            if (Ele[ii]) {
                Force[ii*3    ] = f_ptxc[ii] * force_ramp;
                Force[ii*3 + 1] = f_ptyc[ii] * force_ramp;
                Force[ii*3 + 2] = f_ptzc[ii] * force_ramp;
            }
        }
        __syncthreads();
    }
}


__global__ void force_calc_EMA (
    float *Force,
    double *Force_old, 
    int num_atom, 
    int num_q, 
    float *f_ptxc, 
    float *f_ptyc, 
    float *f_ptzc, 
    int num_atom2, 
    int num_q2, 
    int *Ele,
    double EMA_norm,
    float force_ramp) {
    // Do column tree sum of f_ptxc for f_ptx for every atom, then assign threadIdx.x == 0 (3 * num_atoms) to Force. Force is num_atom * 3. 
    if (blockIdx.x >= num_atom) return;
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        for (int stride = num_q2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                f_ptxc[ii + iAccum * num_atom2] += f_ptxc[ii + iAccum * num_atom2 + stride * num_atom2];
                f_ptyc[ii + iAccum * num_atom2] += f_ptyc[ii + iAccum * num_atom2 + stride * num_atom2];
                f_ptzc[ii + iAccum * num_atom2] += f_ptzc[ii + iAccum * num_atom2 + stride * num_atom2];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            if (Ele[ii]) {
                Force_old[ii*3    ] *= (EMA_norm - 1.0); 
                Force_old[ii*3    ] -= (double)f_ptxc[ii];
                Force_old[ii*3    ] /= EMA_norm;
                Force_old[ii*3 + 1] *= (EMA_norm - 1.0); 
                Force_old[ii*3 + 1] -= (double)f_ptyc[ii];
                Force_old[ii*3 + 1] /= EMA_norm;
                Force_old[ii*3 + 2] *= (EMA_norm - 1.0); 
                Force_old[ii*3 + 2] -= (double)f_ptzc[ii];
                Force_old[ii*3 + 2] /= EMA_norm;
                Force[ii*3    ] = (float)Force_old[ii*3    ] * force_ramp;
                Force[ii*3 + 1] = (float)Force_old[ii*3 + 1] * force_ramp;
                Force[ii*3 + 2] = (float)Force_old[ii*3 + 2] * force_ramp;
            }
        }
        __syncthreads();
    }
}
