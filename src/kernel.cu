#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "env_param.hh"
//#include "scat_param.hh"
#include "mol_param.hh"
#include "WaasKirf.hh"
#define PI 3.14159265359
//#include <hip/hip_fp16.h>


__global__ void dist_calc (
    float *coord, 
    int *close_num,
    int *close_flag, 
    int *close_idx, 
    int num_atom, 
    int num_atom2) {

    // close_flag is a 1024 x num_atom2 int matrix initialized to 0.
    // close_idx: A num_atom x 200 int matrix, row i of which only the first close_num[i] elements are defined. (Otherwise it's -1). 
    __shared__ float x_ref, y_ref, z_ref;
    __shared__ int idz;
    __shared__ int temp[2048];
    // Calc distance
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        if (threadIdx.x == 0) {
            x_ref = coord[3*ii  ];
            y_ref = coord[3*ii+1];
            z_ref = coord[3*ii+2];
        }
        int idy = ii % gridDim.x; // This will be what row of close_flag this block is putting its value in.
        __syncthreads();
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            float r2t = (coord[3*jj  ] - x_ref) * (coord[3*jj  ] - x_ref) + 
                        (coord[3*jj+1] - y_ref) * (coord[3*jj+1] - y_ref) + 
                        (coord[3*jj+2] - z_ref) * (coord[3*jj+2] - z_ref); 
 
            if (r2t < 34.0) {
                close_flag[idy*num_atom2+jj] = 1; // roughly 2 A + 2 A vdW + 2 * 1.8 A probe
            } else { 
                close_flag[idy*num_atom2+jj] = 0;
            }
            if (ii == jj) close_flag[idy*num_atom2+jj] = 0;
        }
        __syncthreads();
        // Do pre scan
        idz = 0;
        int temp_sum = 0;
        for (int jj = threadIdx.x; jj < num_atom2; jj += 2 * blockDim.x) {
            int idx = jj % blockDim.x; 
            int offset = 1;
            temp[2 * idx]     = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx];
            temp[2 * idx + 1] = close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1];
            for (int d = 2 * blockDim.x>>1; d > 0; d >>= 1) { // up-sweep
                __syncthreads();
                if (idx < d) {
                    int ai = offset * (2 * idx + 1) - 1;
                    int bi = offset * (2 * idx + 2) - 1;
                    temp[bi] += temp[ai];
                }
                offset *= 2;
            }
            __syncthreads();
            temp_sum = close_num[ii];
            __syncthreads();
            if (idx == 0) {
                close_num[ii] += temp[2 * blockDim.x - 1]; // log the total number of 1's in this blockDim
                temp[2 * blockDim.x - 1] = 0;
            }
            __syncthreads();
            for (int d = 1; d < blockDim.x * 2; d *= 2) { //down-sweep
                offset >>= 1;
                __syncthreads();
                if (idx < d) {
                    int ai = offset * (2 * idx + 1) - 1;
                    int bi = offset * (2 * idx + 2) - 1;
                    int t    = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
        
            __syncthreads();
        
            // Finally assign the indices
            if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx] == 1) {
                close_idx[ii * 1024 + temp[2*idx] + temp_sum] = 2 * idx + 2 * blockDim.x * idz;
            }
            if (close_flag[idy * num_atom2 + 2 * blockDim.x * idz + 2 * idx + 1] == 1) {
                close_idx[ii * 1024 + temp[2*idx+1] + temp_sum] = 2*idx+1 + 2 * blockDim.x * idz;
            }
            idz++;
            __syncthreads();
        }
    }
}


__global__ void __launch_bounds__(512,4) surf_calc (
    float *coord, 
    int *Ele, 
    int *close_num, 
    int *close_idx, 
    float *vdW, 
    int num_atom, 
    int num_atom2, 
    int num_raster, 
    float sol_s, 
    float *V) {

    // num_raster should be a number of 2^n. 
    // sol_s is solvent radius (default = 1.8 A)
    __shared__ float vdW_s; // vdW radius of the center atom
    __shared__ int pts[512]; // All spherical raster points
    __shared__ float L, r;
    
    if (blockIdx.x >= num_atom) return;
    L = sqrt(num_raster * PI);
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        int atom1t = Ele[ii];
        vdW_s = vdW[atom1t];
        r = vdW_s + sol_s;
        for (int jj = threadIdx.x; jj < num_raster; jj += blockDim.x) {
            int pt = 1;
            
            float h = 1.0 - (2.0 * (float)jj + 1.0) / (float)num_raster;
            float p = acos(h);
            float t = L * p; 
            float xu = sin(p) * cos(t);
            float yu = sin(p) * sin(t);
            float zu = cos(p);
            // vdW points
            float x = vdW_s * xu + coord[3*ii];
            float y = vdW_s * yu + coord[3*ii+1];
            float z = vdW_s * zu + coord[3*ii+2];
            // Solvent center
            float x2 = r * xu + coord[3*ii];
            float y2 = r * yu + coord[3*ii+1];
            float z2 = r * zu + coord[3*ii+2];
            for (int kk = 0; kk < close_num[ii]; kk++) {
                int atom2i = close_idx[ii * 1024 + kk];
                int atom2t = Ele[atom2i];
                float dx = (x - coord[3*atom2i]);
                float dy = (y - coord[3*atom2i+1]);
                float dz = (z - coord[3*atom2i+2]);
                float dx2 = (x2 - coord[3*atom2i]);
                float dy2 = (y2 - coord[3*atom2i+1]);
                float dz2 = (z2 - coord[3*atom2i+2]);
                float dr2 = dx * dx + dy * dy + dz * dz; 
                float dr22 = dx2 * dx2 + dy2 * dy2 + dz2 * dz2;
                // vdW points must not cross into other atom
                if (dr2 < vdW[atom2t] * vdW[atom2t]) pt = 0; //pts[jj] = 0;
                // solvent center has to be far enough
                if (dr22 < (vdW[atom2t]+sol_s) * (vdW[atom2t]+sol_s)) pt = 0; //pts[jj] = 0;
                
            }
            pts[jj] = pt;
        }
        // Sum pts == 1, calc surf area and assign to V[ii]
        for (int stride = num_raster / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                pts[iAccum] += pts[stride + iAccum];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            V[ii] = (float)pts[0]/(float)num_raster;// * 4.0 * r * r * PI ;
        }
    }
}


__global__ void sum_V (
    float *V, 
    float *V_s, 
    int num_atom, 
    int num_atom2, 
    int *Ele,
    float sol_s, 
    float *vdW) {

    for (int ii = threadIdx.x; ii < num_atom2; ii += blockDim.x) {
        if (ii < num_atom) {
            int atomi = Ele[ii];
            if (atomi > 5) atomi = 0;
            V_s[ii] = V[ii] * 4.0 * PI * (vdW[atomi]+sol_s) * (vdW[atomi]+sol_s);
        } else {
            V_s[ii] = 0.0;
        }
    }
    for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
            V_s[iAccum] += V_s[stride + iAccum];
        }
    }
    __syncthreads();
    if (threadIdx.x == 0) printf("Convex contact area = %.3f A^2.\n", V_s[0]);
}

__global__ void FF_calc (
    float *q, 
    float *WK, 
    float *vdW, 
    int num_q, 
    int num_ele, 
    float c1, 
    float r_m, 
    float *FF_table,
    float rho) {

    // Calculate the non-SASA part of form factors per element

    __shared__ float q_pt, q_WK, C1, expC1;
    __shared__ float FF_pt[99]; // num_ele + 1, the last one for water.
    __shared__ float vdW_s[99];
    __shared__ float WK_s[1078]; 
    __shared__ float C1_PI_43_rho;
    if (blockIdx.x >= num_q) return; // out of q range
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        q_pt = q[ii];
        q_WK = q_pt / 4.0 / PI;
        // FoXS C1 term
        expC1 = -powf(4.0 * PI / 3.0, 1.5) * q_WK * q_WK * r_m * r_m * (c1 * c1 - 1.0) / 4.0 / PI;
        C1 = powf(c1,3) * exp(expC1);
        C1_PI_43_rho = C1 * PI * 4.0 / 3.0 * rho;
        for (int jj = threadIdx.x; jj < 11 * num_ele; jj += blockDim.x) {
            WK_s[jj] = WK[jj];
        } // Copy WK to shared memory for faster access
        __syncthreads();

        // Calculate Form factor for this block (or q vector)
        for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
            vdW_s[jj] = vdW[jj];
            if (jj == num_ele) {
                // water
                FF_pt[jj] = WK_s[7*11+5];  // Oxygen
                FF_pt[jj] += 2.0 * WK_s[5];  // Hydrogen
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * powf(4.0/3.0*PI, 2.0/3.0) * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);  // Water vdW_s
                for (int kk = 0; kk < 5; kk ++) {
                    FF_pt[jj] += WK_s[7*11+kk] * exp(-WK_s[7*11+kk+6] * q_WK * q_WK); // Oxygen
                    FF_pt[jj] += WK_s[kk] * exp(-WK_s[kk+6] * q_WK * q_WK) * 2.0; // Hydrogen
                }
            } else { 
                FF_pt[jj] = WK_s[jj*11+5];
                // The part is for excluded volume
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * powf(4.0/3.0*PI, 2.0/3.0) * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);  // Water vdW_s
                //FF_pt[jj] -= C1_PI_43_rho * powf(vdW_s[jj],3.0) * exp(-PI * vdW_s[jj] * vdW_s[jj] * q_WK * q_WK);
                for (int kk = 0; kk < 5; kk++) {
                    FF_pt[jj] += WK_s[jj*11+kk] * exp(-WK_s[jj*11+kk+6] * q_WK * q_WK); 
                }
            }
            FF_table[ii*(num_ele+1)+jj] = FF_pt[jj];
        }
    }
}


__global__ void create_FF_full_FoXS (
    float *FF_table, 
    float *V, 
    float c2, 
    int *Ele, 
    float *FF_full, 
    int num_q, 
    int num_ele, 
    int num_atom, 
    int num_atom2) {

    // Add on SASA for each atom

    __shared__ float FF_pt[99];
    float hydration;
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        // Get form factor for this block (or q vector)
        if (ii < num_q) {
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
            }
        }
        __syncthreads();
        
        // In FoXS since c2 remains the same for all elements it is reduced to one value.
        hydration = c2 * FF_pt[num_ele];
        
        // Calculate atomic form factor for this q
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            int atomt = Ele[jj];
            FF_full[ii*num_atom2 + jj] = FF_pt[atomt];
            FF_full[ii*num_atom2 + jj] += hydration * V[jj];
        }
    }
}

__global__ void __launch_bounds__(1024,2) pure_scat_calc (
    float *coord, 
    int *Ele,
    float *q,
    float *S_calc, 
    int num_atom,   
    int num_q,     
    int num_ele,   
    float *S_calcc, 
    int num_atom2,
    float *FF_full) {

    float q_pt;

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {

        q_pt = q[ii];

        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // for every atom jj
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            float S_calccs = 0.0;
            for (int kk = 0; kk < num_atom; kk++) {
                // for every atom kk
                float FF_kj = FF_full[ii * num_atom2 + jj] * FF_full[ii *num_atom2 + kk];
                if (q_pt == 0.0 || kk == jj) {
                    S_calccs += FF_kj;
                } else {
                    float dx = atom1x - coord[3*kk+0];
                    float dy = atom1y - coord[3*kk+1];
                    float dz = atom1z - coord[3*kk+2];
                    float r = sqrt(dx*dx+dy*dy+dz*dz);
                    float qr = q_pt * r; 
                    float sqr = sin(qr) / qr;
                    S_calccs += FF_kj * sqr;
                }
            }
            S_calcc[ii*num_atom2+jj] = S_calccs;
        }
        
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        
        S_calc[ii] = S_calcc[ii * num_atom2];
        __syncthreads();


    }
}



