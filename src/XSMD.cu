#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include "kernel.cu"
#include "XSMD.hh"
#include "mol_param.hh"
#include "env_param.hh"
#include "scat_param.hh"
#include "WaasKirf.hh"
#define PI 3.14159265359 

void XSMD_calc (float *coord, float *Force, double *Force_old, double *S_old, int frame_num, double *EMA_norm, int restart) {
if (frame_num % delta_t == 0) {
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);

    printf("Frame %d, doing things ...\n", frame_num);
    // In this code pointers with d_ are device pointers. 

    // Declare local pointers //
    // The calculated scattering pattern for this snapshot.
    //float *S_calc;
    double *S_calc;

    // Declare cuda pointers //
    float *d_coord;          // Coordinates 3 x num_atom
    float *d_Force;          // Force 3 x num_atom
    double *d_Force_old;

    int   *d_Ele;            // Element list.

    float *d_q_S_ref_dS;     /* q vector, reference scattering pattern, and 
                                measured difference pattern to fit.
                                Since they are of same size they're grouped */
    float *d_sigma2;         // Sigma square (standard error of mean) for the target diff pattern.
                                
    float *d_Aq;             // Prefactor for each q
    //float *d_S_calc;         // Calculated scattering curve
    double *d_S_calc;         // For EMA method, use double

    float *d_S_calcc,        // Some intermediate matrices
          *d_f_ptxc, 
          *d_f_ptyc, 
          *d_f_ptzc;
    
    float *d_V,              // Exposed surf area (in num of dots) 
          *d_V_s;            // Exposed surf area (in real A^2)

    float *d_WK;             // Waasmaier-Kirfel parameters 

    int   *d_close_flag,     // Flags for atoms close to an atom
          *d_close_num,      // Num of atoms close to an atom
          *d_close_idx;      // Their atomic index
 
    float *d_vdW;            // van der Waals radii

    float *d_FF_table,       // Form factors for each atom type at each q
          *d_FF_full;        /* Form factors for each atom at each q, 
                                considering the SASA an atom has. */
    // If using HyPred mode, then an array of c2 is needed. //
    float *d_c2;
    
    // If using surface gradient 
    float *d_surf_grad;


    // Compute the exponential moving average normalization constant.
    // Here this final 500.0 is to say we average over 500 snapshots,
    // each snapshot taken every 1000 steps (the first if statement of this kernel).
    // So we have tau = 1.0 ns for exponential averaging.
    double *d_S_old;
    *EMA_norm = *EMA_norm * exp(-(float)delta_t/(float)tau) + 1.0;
    printf("Currently EMA_Norm is %.3f\n",*EMA_norm);
    float force_ramp;
    if (frame_num < tau) {
        force_ramp = 0.0;
    } else if (frame_num > 2 * tau) {
        force_ramp = 1.0;
    } else {
        force_ramp = (1 - cos(PI * ((float)frame_num - (float)tau) / (float)tau)) / 2.0;
    }
    if (restart == 1) {
        if (frame_num == 0) printf("Restarting a run, setting force_ramp to 1\n");
        force_ramp = 1.0;
    }
    //force_ramp = 1.0;
    printf("Currently force_ramp is %.3f\n",force_ramp);
    

    
    // set various memory chunk sizes
    int size_coord       = 3 * num_atom * sizeof(float);
    int size_double_coord= 3 * num_atom * sizeof(double);
    int size_atom        = num_atom * sizeof(int);
    int size_atom2       = num_atom2 * sizeof(int);
    int size_atom2f      = num_atom2 * sizeof(float);
    int size_atom2xatom2 = 1024 * num_atom2 * sizeof(int); // For d_close_flag
    int size_q           = num_q * sizeof(float); 
    int size_double_q    = num_q * sizeof(double);
    int size_qxatom2     = num_q2 * num_atom2 * sizeof(float);
    int size_FF_table    = (num_ele + 1) * num_q * sizeof(float); // +1 for solvent
    int size_WK          = 11 * num_ele * sizeof(float);
    int size_vdW         = (num_ele + 1) * sizeof(float); // +1 for solvent
    int size_c2          = 10 * sizeof(float); // Only for HyPred

    // Allocate local memories
    //S_calc = (float *)malloc(size_q);
    S_calc = (double *)malloc(size_double_q);

    // Allocate cuda memories
    hipMalloc((void **)&d_Aq,         size_q);
    hipMalloc((void **)&d_coord,      size_coord); // 40 KB
    hipMalloc((void **)&d_Force,      size_coord); // 40 KB
    hipMalloc((void **)&d_Force_old,  size_double_coord); // 40 KB
    hipMalloc((void **)&d_Ele,        size_atom);
    hipMalloc((void **)&d_q_S_ref_dS, 3 * size_q);
    hipMalloc((void **)&d_sigma2,     size_q);
    //hipMalloc((void **)&d_S_calc,     size_q); // Will be computed on GPU
    hipMalloc((void **)&d_S_calc,     size_double_q); // For EMA method, use double precision
    hipMalloc((void **)&d_f_ptxc,     size_qxatom2);
    hipMalloc((void **)&d_f_ptyc,     size_qxatom2);
    hipMalloc((void **)&d_f_ptzc,     size_qxatom2);
    hipMalloc((void **)&d_S_calcc,    size_qxatom2);
    hipMalloc((void **)&d_V,          size_atom2f);
    hipMalloc((void **)&d_V_s,        size_atom2f);
    hipMalloc((void **)&d_close_flag, size_atom2xatom2);
    hipMalloc((void **)&d_close_num,  size_atom2);
    hipMalloc((void **)&d_close_idx,  size_atom2xatom2);
    hipMalloc((void **)&d_vdW,        size_vdW);
    hipMalloc((void **)&d_FF_table,   size_FF_table);
    hipMalloc((void **)&d_FF_full,    size_qxatom2);
    hipMalloc((void **)&d_WK,         size_WK);
    hipMalloc((void **)&d_c2,         size_c2); // Only for HyPred
    hipMalloc((void **)&d_S_old,      size_double_q); // For EMA
    hipMalloc((void **)&d_surf_grad, size_coord); // For surface gradient

    // Initialize some matrices
    hipMemset(d_close_flag, 0,   size_qxatom2);
    hipMemset(d_Force,      0.0, size_coord);
    hipMemset(d_Aq,         0.0, size_q);
    //hipMemset(d_S_calc,     0.0, size_q);
    hipMemset(d_S_calc,     0.0, size_double_q); //For EMA method, use double precision
    hipMemset(d_f_ptxc,     0.0, size_qxatom2);
    hipMemset(d_f_ptyc,     0.0, size_qxatom2);   
    hipMemset(d_f_ptzc,     0.0, size_qxatom2);
    hipMemset(d_S_calcc,    0.0, size_qxatom2);
    hipMemset(d_close_num,  0,   size_atom2);
    hipMemset(d_close_idx,  0,   size_atom2xatom2);
    hipMemset(d_surf_grad,  0.0, size_coord);
    hipMemset(d_FF_full,    0.0, size_qxatom2);

    // Copy necessary data
    hipMemcpy(d_coord,      coord,      size_coord, hipMemcpyHostToDevice);
    hipMemcpy(d_Force_old,  Force_old,  size_double_coord, hipMemcpyHostToDevice);
    hipMemcpy(d_vdW,        vdW,        size_vdW,   hipMemcpyHostToDevice);
    hipMemcpy(d_Ele,        Ele,        size_atom,  hipMemcpyHostToDevice);
    hipMemcpy(d_q_S_ref_dS, q_S_ref_dS, 3 * size_q, hipMemcpyHostToDevice);
    hipMemcpy(d_sigma2,     dS_err,     size_q,     hipMemcpyHostToDevice);
    hipMemcpy(d_WK,         WK,         size_WK,    hipMemcpyHostToDevice);
    // Only for HyPred
    hipMemcpy(d_c2,         c2_H,       size_c2,    hipMemcpyHostToDevice);
    hipMemcpy(d_S_old,      S_old,      size_double_q, hipMemcpyHostToDevice);

    //float sigma2 = 1.0;
    float alpha = 1.0;
    float offset = 0.2;
 
    dist_calc<<<1024, 1024>>>(
        d_coord, 
        d_close_num, 
        d_close_flag,
        d_close_idx, 
        num_atom,
        num_atom2); 

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    surf_calc<<<1024,512>>>(
        d_coord, 
        d_Ele, 
        d_close_num, 
        d_close_idx, 
        d_vdW, 
        num_atom, 
        num_atom2, 
        num_raster, 
        sol_s, 
        d_V);

    /*surf_calc_surf_grad<<<1024,512>>>(
        d_coord, 
        d_Ele, 
        d_close_num, 
        d_close_idx, 
        d_vdW, 
        num_atom, 
        num_atom2, 
        num_raster, 
        sol_s, 
        d_V,
        d_surf_grad,
        offset);
    */
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    sum_V<<<1,1024>>>(
        d_V, 
        d_V_s, 
        num_atom, 
        num_atom2, 
        d_Ele, 
        d_vdW);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    FF_calc<<<320, 32>>>(
        d_q_S_ref_dS, 
        d_WK, 
        d_vdW, 
        num_q, 
        num_ele, 
        c1, 
        r_m, 
        d_FF_table,
        rho);
/*
    create_FF_full_HyPred<<<320, 1024>>>(
        d_FF_table, 
        d_V,
        c2, 
        d_c2, 
        d_Ele, 
        d_FF_full, 
        num_q, 
        num_ele, 
        num_atom, 
        num_atom2);
*/
    create_FF_full_FoXS<<<320, 1024>>>(
        d_FF_table, 
        d_V,
        c2, 
        d_Ele, 
        d_FF_full, 
        num_q, 
        num_ele, 
        num_atom, 
        num_atom2);

/*
    create_FF_full_FoXS_surf_grad<<<320, 1024>>>(
        d_FF_table, 
        d_V,
        c2, 
        d_Ele, 
        d_FF_full,
        d_surf_grad, 
        num_q, 
        num_ele, 
        num_atom, 
        num_atom2);
*/
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }

/*    scat_calc<<<320, 1024>>>(
        d_coord, 
        d_Ele,
        d_q_S_ref_dS, 
        d_S_calc, 
        num_atom,  
        num_q,     
        num_ele,  
        d_Aq, 
        alpha,    
        k_chi,     
        d_sigma2,    
        d_f_ptxc, 
        d_f_ptyc, 
        d_f_ptzc, 
        d_S_calcc, 
        num_atom2, 
        d_FF_full);
*/
/*
    scat_calc_surf_grad<<<320, 1024>>>(
        d_coord, 
        d_Ele,
        d_q_S_ref_dS, 
        d_S_calc, 
        num_atom,  
        num_q,     
        num_ele,  
        d_Aq, 
        alpha,    
        k_chi,     
        d_sigma2,    
        d_f_ptxc, 
        d_f_ptyc, 
        d_f_ptzc, 
        d_S_calcc, 
        num_atom2, 
        d_surf_grad,
        d_FF_full);
*/

    scat_calc_EMA<<<320, 1024>>>(
        d_coord, 
        d_Ele,
        d_q_S_ref_dS, 
        d_S_calc, 
        num_atom,  
        num_q,     
        num_ele,  
        d_Aq, 
        alpha,    
        k_chi,     
        d_sigma2,    
        d_f_ptxc, 
        d_f_ptyc, 
        d_f_ptzc, 
        d_S_calcc, 
        num_atom2, 
        d_FF_full,
        d_S_old,
        *EMA_norm
        );


    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    //hipMemcpyAsync(S_calc, d_S_calc, size_q,     hipMemcpyDeviceToHost);
    hipMemcpyAsync(S_calc, d_S_calc, size_double_q,     hipMemcpyDeviceToHost);

    force_calc<<<1024, 512>>>(
        d_Force,
        num_atom, 
        num_q, 
        d_f_ptxc, 
        d_f_ptyc, 
        d_f_ptzc, 
        num_atom2, 
        num_q2, 
        d_Ele,
        force_ramp);
/*    force_calc_EMA<<<1024, 512>>>(
        d_Force,
        d_Force_old,
        num_atom, 
        num_q, 
        d_f_ptxc, 
        d_f_ptyc, 
        d_f_ptzc, 
        num_atom2, 
        num_q2, 
        d_Ele,
        *EMA_norm,
        force_ramp);
*/
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }

    hipMemcpy(Force,      d_Force,      size_coord, hipMemcpyDeviceToHost);
    //hipMemcpy(Force_old,  d_Force_old,  size_double_coord, hipMemcpyDeviceToHost);
    hipMemcpy(S_old,      d_S_old,      size_double_q, hipMemcpyDeviceToHost);

    float chi = 0.0;
    float chi2 = 0.0;
    float chi_ref = 0.0;
    if (frame_num % 1000 == 0) printf("S_calc: ");
    //if (frame_num % 1000 == 0) printf("S_old: ");
    for (int ii = 0; ii < num_q; ii++) {
        //chi = q_S_ref_dS[ii+2*num_q] - ((float)S_old[ii] - q_S_ref_dS[ii+num_q]);
        chi = q_S_ref_dS[ii+2*num_q] - ((float)S_calc[ii] - q_S_ref_dS[ii+num_q]);
        chi2 += chi * chi / dS_err[ii];
        chi_ref+= q_S_ref_dS[ii+2*num_q] * q_S_ref_dS[ii+2*num_q] / dS_err[ii];
        if (frame_num % 1000 == 0) printf("%.3f, ", S_calc[ii]);
    }
    printf("\nchi square is %.5e ( %.3f % )\n", chi2, chi2 / chi_ref * 100);

    if (frame_num % 1000 == 0) {
        /*printf("Force vectors: \n");
        for (int ii = 0; ii < num_atom; ii++) {
            printf("%8.5f %8.5f %8.5f\n", Force[3*ii+0], Force[3*ii+1], Force[3*ii+2]);
        }*/
        /*printf("Force_old vectors: \n");
        for (int ii = 0; ii < num_atom; ii++) {
            printf("%8.5f %8.5f %8.5f\n", Force_old[3*ii+0], Force_old[3*ii+1], Force_old[3*ii+2]);
        }*/
    } 
    hipFree(d_coord); 
    hipFree(d_Force); 
    hipFree(d_Force_old); 
    hipFree(d_Ele); 
    hipFree(d_q_S_ref_dS);
    hipFree(d_sigma2); 
    hipFree(d_Aq);
    hipFree(d_S_calc); 
    hipFree(d_S_calcc); 
    hipFree(d_f_ptxc); hipFree(d_f_ptyc); hipFree(d_f_ptzc);
    hipFree(d_V); hipFree(d_V_s); 
    hipFree(d_WK);
    hipFree(d_close_flag); hipFree(d_close_num); hipFree(d_close_idx);
    hipFree(d_vdW);
    hipFree(d_FF_table); hipFree(d_FF_full);
    hipFree(d_S_old);
    hipFree(d_c2);
    hipFree(d_surf_grad);
    free(S_calc);

    gettimeofday(&tv2, NULL);
    double time_in_mill = 
         (tv2.tv_sec - tv1.tv_sec) * 1000.0 + (tv2.tv_usec - tv1.tv_usec) / 1000.0 ;
    printf("Time elapsed = %.3f ms.\n", time_in_mill);
}
}
